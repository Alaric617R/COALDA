
#include <hip/hip_runtime.h>
#include <cstdio>

struct pixel
{
  int r;
  int g;
  int b;

  bool operator==(const pixel &rhs)
  {
    return (r == rhs.r) && (g == rhs.g) && (b == rhs.b);
  };
  bool operator!=(const pixel &rhs)
  {
    return !(*this == rhs);
  }
};

// Commonly used, intuitive but uncoalesced
__global__ void rgb_copy_interleaved(pixel *pixel_dst, pixel *pixel_src)
{
  pixel_dst[threadIdx.x].r = pixel_src[threadIdx.x].r;
  pixel_dst[threadIdx.x].g = pixel_src[threadIdx.x].g;
  pixel_dst[threadIdx.x].b = pixel_src[threadIdx.x].b;
};

int main()
{
  pixel host_pixel_src[32];
  pixel host_pixel_res[32];
  pixel *device_pixel_src;
  pixel *device_pixel_cpy;

  srand(1);
  for (int i = 0; i < 32; i++)
  {
    host_pixel_src[i] = {rand() % 256, rand() % 256, rand() % 256};
  }

  printf("Host data initialized:\n");
  for (int i = 0; i < 32; i++)
  {
    printf("%d %d %d %d\n", i, host_pixel_src[i].r, host_pixel_src[i].g, host_pixel_src[i].b);
  }

  hipMalloc(&device_pixel_src, 32 * sizeof(pixel));
  hipMalloc(&device_pixel_cpy, 32 * sizeof(pixel));

  hipMemcpy(device_pixel_src, host_pixel_src, 32 * sizeof(pixel), hipMemcpyHostToDevice);

  dim3 dimGrid = dim3(1, 1, 1);
  dim3 dimBlock = dim3(32, 1, 1);

  rgb_copy_interleaved<<<dimGrid, dimBlock, 32 * sizeof(pixel)>>>(device_pixel_cpy, device_pixel_src);
  hipDeviceSynchronize();

  hipMemcpy(host_pixel_res, device_pixel_cpy, 32 * sizeof(pixel), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  bool success = true;
  printf("Data after device internal copy:\n");
  for (int i = 0; i < 32; i++)
  {
    printf("%d %d %d %d\n", i, host_pixel_res[i].r, host_pixel_res[i].g, host_pixel_res[i].b);
    if (host_pixel_res[i] != host_pixel_src[i]) {
      success = false;
      printf("Error on pixel %d\n", i);
    }
  }
  if (success) {
    printf("All matched!\n");
  }

  hipFree(device_pixel_src);
  hipFree(device_pixel_cpy);

  return 0;
}