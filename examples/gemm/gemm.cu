
#include <hip/hip_runtime.h>
/*
    A: m x n
    B: n x p
    C: m x p
    For simplicity, assume dimensions are multiple of TILE_WIDTH
*/

 #define TILE_WIDTH 16

__global__ void gemm(const float* A, const float* B, float* C, int m, int n, int p) {
  __shared__ float As[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float sum = 0;
  for (int k = 0; k < n; k+= TILE_WIDTH) {
    As[threadIdx.y][threadIdx.x] = A[row * n + (k + threadIdx.x)];
    Bs[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * p + col];
    __syncthreads();
    for (int i = 0; i < TILE_WIDTH; i++) {
      sum += As[threadIdx.y][i] * Bs[i][threadIdx.x];
    }
    __syncthreads();
  }
  C[row * p + col] = sum;
}