#include "hip/hip_runtime.h"
__global__ void rgb_increase_brightness(int *pixel_dst, int *pixel_src,
                                        int size, float factor) {
  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
    TODO: uncoalesced, with slightly complicated pattern but still a continuous
    region of memory
  */
  pixel_smem_src[3 * local_tid + 0] = pixel_src[3 * global_tid + 0];  // r
  pixel_smem_src[3 * local_tid + 1] = pixel_src[3 * global_tid + 1];  // g
  pixel_smem_src[3 * local_tid + 2] = pixel_src[3 * global_tid + 2];  // b

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 0])));
  pixel_smem_dst[3 * local_tid + 1] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 1])));
  pixel_smem_dst[3 * local_tid + 2] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 2])));

  /*
    Write result to destination
    TODO: uncoalesced
  */
  pixel_dst[3 * global_tid + 0] = pixel_smem_dst[3 * local_tid + 0];  // r
  pixel_dst[3 * global_tid + 1] = pixel_smem_dst[3 * local_tid + 1];  // g
  pixel_dst[3 * global_tid + 2] = pixel_smem_dst[3 * local_tid + 2];  // b
};

__global__ void rgb_increase_brightness_coalesced(int *pixel_dst,
                                                  int *pixel_src, int size,
                                                  float factor) {
  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
    TODO: uncoalesced, with slightly complicated pattern but still a continuous
    region of memory
  */
  pixel_smem_src[local_tid + 0 * blockDim.x] =
      pixel_src[global_tid + 0 * blockDim.x];  // r
  pixel_smem_src[local_tid + 1 * blockDim.x] =
      pixel_src[global_tid + 1 * blockDim.x];  // g
  pixel_smem_src[local_tid + 2 * blockDim.x] =
      pixel_src[global_tid + 2 * blockDim.x];  // b

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 0])));
  pixel_smem_dst[3 * local_tid + 1] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 1])));
  pixel_smem_dst[3 * local_tid + 2] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 2])));

  /*
    Write result to destination
    TODO: uncoalesced
  */
  pixel_dst[global_tid + 0 * blockDim.x] =
      pixel_smem_dst[local_tid + 0 * blockDim.x];  // r
  pixel_dst[global_tid + 1 * blockDim.x] =
      pixel_smem_dst[local_tid + 1 * blockDim.x];  // g
  pixel_dst[global_tid + 2 * blockDim.x] =
      pixel_smem_dst[local_tid + 2 * blockDim.x];  // b
};
