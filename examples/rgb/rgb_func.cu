#include "hip/hip_runtime.h"
__global__ void rgb_increase_brightness(int *pixel_dst, int *pixel_src,
                                        int size, float factor) {
  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
    TODO: uncoalesced, with slightly complicated pattern but still a continuous
    region of memory
  */
  pixel_smem_src[3 * local_tid + 0] = pixel_src[3 * global_tid + 0];  // r
  pixel_smem_src[3 * local_tid + 1] = pixel_src[3 * global_tid + 1];  // g
  pixel_smem_src[3 * local_tid + 2] = pixel_src[3 * global_tid + 2];  // b

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] =
      min(255, (int)(factor + 1.0f) * (pixel_smem_src[3 * local_tid + 0]));
  pixel_smem_dst[3 * local_tid + 1] =
      min(255, (int)(factor + 1.0f) * (pixel_smem_src[3 * local_tid + 1]));
  pixel_smem_dst[3 * local_tid + 2] =
      min(255, (int)(factor + 1.0f) * (pixel_smem_src[3 * local_tid + 2]));

  /*
    Write result to destination
    TODO: uncoalesced
  */
  pixel_dst[3 * global_tid + 0] = pixel_smem_dst[3 * local_tid + 0];  // r
  pixel_dst[3 * global_tid + 1] = pixel_smem_dst[3 * local_tid + 1];  // g
  pixel_dst[3 * global_tid + 2] = pixel_smem_dst[3 * local_tid + 2];  // b
};
