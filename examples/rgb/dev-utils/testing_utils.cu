#include <stdio.h>

#include "rgb.cu"

void test_rgb_array() {
  const dim3 dimGrid = dim3(1, 1, 1);
  const dim3 dimBlock = dim3(TILE_WIDTH, 1, 1);
  int num_pixels = dimGrid.x * dimBlock.x;
  int host_pixel_src[3 * num_pixels];
  int host_pixel_res[3 * num_pixels];
  int *device_pixel_src;
  int *device_pixel_cpy;
  srand(1);
  for (int i = 0; i < num_pixels; i++) {
    host_pixel_src[3 * i + 0] = rand() % 256;
    host_pixel_src[3 * i + 1] = rand() % 256;
    host_pixel_src[3 * i + 2] = rand() % 256;
  }
  printf("Host data initialized:\n");
  for (int i = 0; i < num_pixels; i++) {
    printf("%d %d %d %d\n", i, host_pixel_src[3 * i + 0],
           host_pixel_src[3 * i + 1], host_pixel_src[3 * i + 2]);
  }
  hipMalloc(&device_pixel_src, 3 * num_pixels * sizeof(int));
  hipMalloc(&device_pixel_cpy, 3 * num_pixels * sizeof(int));
  hipMemcpy(device_pixel_src, host_pixel_src, 3 * num_pixels * sizeof(int),
             hipMemcpyHostToDevice);

  rgb_copy_array_interleaved<<<dimGrid, dimBlock>>>(device_pixel_cpy,
                                                    device_pixel_src);
  hipDeviceSynchronize();

  hipMemcpy(host_pixel_res, device_pixel_cpy, 3 * num_pixels * sizeof(int),
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  bool success = true;
  printf("Data after device internal copy:\n");
  for (int i = 0; i < num_pixels; i++) {
    printf("%d %d %d %d\n", i, host_pixel_res[3 * i + 0],
           host_pixel_res[3 * i + 1], host_pixel_res[3 * i + 2]);
    if (host_pixel_res[3 * i + 0] != host_pixel_src[3 * i + 0] ||
        host_pixel_res[3 * i + 1] != host_pixel_src[3 * i + 1] ||
        host_pixel_res[3 * i + 2] != host_pixel_src[3 * i + 2]) {
      success = false;
      printf("Error on pixel %d\n", i);
    }
  }
  if (success) {
    printf("All matched!\n");
  }

  hipFree(device_pixel_src);
  hipFree(device_pixel_cpy);
}

void test_increase_brightness() {
  int width, height;
  int *host_pixel_src = read_ppm("images/1.ppm", width, height);
  long long total_pixel = width * height;
  int num_block = (total_pixel + TILE_WIDTH - 1) / TILE_WIDTH;
  const dim3 dimGrid = dim3(num_block, 1, 1);
  const dim3 dimBlock = dim3(TILE_WIDTH, 1, 1);
  int num_pixels = width * height;
  int host_pixel_res[3 * num_pixels];
  int *device_pixel_src;
  int *device_pixel_cpy;
  hipMalloc(&device_pixel_src, 3 * num_pixels * sizeof(int));
  hipMalloc(&device_pixel_cpy, 3 * num_pixels * sizeof(int));
  hipMemcpy(device_pixel_src, host_pixel_src, 3 * num_pixels * sizeof(int),
             hipMemcpyHostToDevice);
  rgb_increase_brightness<<<dimGrid, dimBlock>>>(
      device_pixel_cpy, device_pixel_src, width * height, 1.8);
  hipDeviceSynchronize();
  hipMemcpy(host_pixel_res, device_pixel_cpy, 3 * num_pixels * sizeof(int),
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  write_ppm("images/1_modified.ppm", host_pixel_res, width, height);
  delete[] host_pixel_src;
  hipFree(device_pixel_src);
  hipFree(device_pixel_cpy);
}

void test_increase_brightness_unroll() {
  int width, height;
  int *host_pixel_src = read_ppm("images/1.ppm", width, height);
  long long total_pixel = width * height / 4;
  int num_block = (total_pixel + TILE_WIDTH - 1) / TILE_WIDTH;
  const dim3 dimGrid = dim3(num_block, 1, 1);
  const dim3 dimBlock = dim3(TILE_WIDTH, 1, 1);
  int num_pixels = width * height;
  int host_pixel_res[4 * 3 * num_pixels];
  int *device_pixel_src;
  int *device_pixel_cpy;
  hipMalloc(&device_pixel_src, 4 * 3 * num_pixels * sizeof(int));
  hipMalloc(&device_pixel_cpy, 4 * 3 * num_pixels * sizeof(int));
  hipMemcpy(device_pixel_src, host_pixel_src, 4 * 3 * num_pixels * sizeof(int),
             hipMemcpyHostToDevice);
  rgb_increase_brightness_coalesced_unroll<<<dimGrid, dimBlock>>>(
      device_pixel_cpy, device_pixel_src, width * height, 1.8);
  hipDeviceSynchronize();
  hipMemcpy(host_pixel_res, device_pixel_cpy, 4 * 3 * num_pixels * sizeof(int),
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  write_ppm("images/1_modified.ppm", host_pixel_res, width, height);
  delete[] host_pixel_src;
  hipFree(device_pixel_src);
  hipFree(device_pixel_cpy);
}