#include <cstdio>

#include "ppm_utils.h"
#include "rgb_pass_ready.cu"

#define TILE_WIDTH 512

void test_increase_brightness_pass_ready() {
  int width, height;
  int *host_pixel_src = read_ppm("images/1.ppm", width, height);
  long long total_pixel = width * height;
  int num_block = (total_pixel + TILE_WIDTH - 1) / TILE_WIDTH;
  const dim3 dimGrid = dim3(num_block, 1, 1);
  const dim3 dimBlock = dim3(TILE_WIDTH, 1, 1);
  int num_pixels = width * height;
  int host_pixel_res[3 * num_pixels];
  int *device_pixel_src;
  int *device_pixel_cpy;
  hipMalloc(&device_pixel_src, 3 * num_pixels * sizeof(int));
  hipMalloc(&device_pixel_cpy, 3 * num_pixels * sizeof(int));
  hipMemcpy(device_pixel_src, host_pixel_src, 3 * num_pixels * sizeof(int),
             hipMemcpyHostToDevice);
  rgb_increase_brightness_pass_ready<<<dimGrid, dimBlock>>>(
      device_pixel_cpy, device_pixel_src, width * height, 1.8);
  hipDeviceSynchronize();
  hipMemcpy(host_pixel_res, device_pixel_cpy, 3 * num_pixels * sizeof(int),
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  write_ppm("images/1_modified.ppm", host_pixel_res, width, height);
  delete[] host_pixel_src;
  hipFree(device_pixel_src);
  hipFree(device_pixel_cpy);
}

int main() {
  test_increase_brightness_pass_ready();
  return 0;
}