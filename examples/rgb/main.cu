#include "hip/hip_runtime.h"
#include <cstdio>

#include "ppm_utils.h"
// #include "rgb_pass_ready.cu"

#define TILE_WIDTH 512

__global__ void rgb_increase_brightness_pass_ready(int *pixel_dst,
                                                   int *pixel_src, int size,
                                                   float factor) {
  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    "Gather": Read input pixels to shared memory
  */
  pixel_smem_src[3 * local_tid + 0] = pixel_src[3 * global_tid + 0];  // r
  __syncthreads();
  pixel_smem_src[3 * local_tid + 1] = pixel_src[3 * global_tid + 1];  // g
  __syncthreads();
  pixel_smem_src[3 * local_tid + 2] = pixel_src[3 * global_tid + 2];  // b
  __syncthreads();

  /*
    Computation, very simple here but can be more complicated
  */
  pixel_smem_dst[3 * local_tid + 0] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 0])));
  pixel_smem_dst[3 * local_tid + 1] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 1])));
  pixel_smem_dst[3 * local_tid + 2] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 2])));
  __syncthreads();

  /*
    "Scatter": Write result to destination
  */
  pixel_dst[3 * global_tid + 0] = pixel_smem_dst[3 * local_tid + 0];  // r
  __syncthreads();
  pixel_dst[3 * global_tid + 1] = pixel_smem_dst[3 * local_tid + 1];  // g
  __syncthreads();
  pixel_dst[3 * global_tid + 2] = pixel_smem_dst[3 * local_tid + 2];  // b
};

void test_increase_brightness_pass_ready() {

  /*
    Declaration and configuration
  */
  int width, height;
  int *host_pixel_src = read_ppm("images/1.ppm", width, height);
  long long total_pixel = width * height;
  int num_block = (total_pixel + TILE_WIDTH - 1) / TILE_WIDTH;
  const dim3 dimGrid = dim3(num_block, 1, 1);
  const dim3 dimBlock = dim3(TILE_WIDTH, 1, 1);
  int num_pixels = width * height;
  int host_pixel_res[3 * num_pixels];
  int *device_pixel_src;
  int *device_pixel_cpy;

  /*
    Allocate host and device memory
  */
  hipMalloc(&device_pixel_src, 3 * num_pixels * sizeof(int));
  hipMalloc(&device_pixel_cpy, 3 * num_pixels * sizeof(int));

  /*
    Data movement and kernel launch
  */
  hipMemcpy(device_pixel_src, host_pixel_src, 3 * num_pixels * sizeof(int),
             hipMemcpyHostToDevice);
  rgb_increase_brightness_pass_ready<<<dimGrid, dimBlock>>>(
      device_pixel_cpy, device_pixel_src, width * height, 1.8);
  hipDeviceSynchronize();
  hipMemcpy(host_pixel_res, device_pixel_cpy, 3 * num_pixels * sizeof(int),
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  /*
    Write result to output file
  */
  write_ppm("images/1_modified.ppm", host_pixel_res, width, height);


  /*
    Free host and device memory
  */
  delete[] host_pixel_src;
  hipFree(device_pixel_src);
  hipFree(device_pixel_cpy);
}

int main() {
  test_increase_brightness_pass_ready();
  return 0;
}