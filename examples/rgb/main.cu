#include "rgb.cu"
#include <cstdio>

int main()
{
  pixel host_pixel[32];
  pixel *device_pixel_src;
  pixel *device_pixel_cpy;

  srand(1);
  for (int i = 0; i < 32; i++)
  {
    host_pixel[i] = {rand() % 256, rand() % 256, rand() % 256};
  }

  printf("Host data initialized:\n");
  for (int i = 0; i < 32; i++)
  {
    printf("%d %d %d %d\n", i, host_pixel[i].r, host_pixel[i].g, host_pixel[i].b);
  }

  hipMalloc(&device_pixel_src, 32 * sizeof(pixel));
  hipMalloc(&device_pixel_src, 32 * sizeof(pixel));

  hipMemcpy(device_pixel_src, host_pixel, 32 * sizeof(pixel),
             hipMemcpyHostToDevice);

  dim3 dimGrid = (32, 1, 1);
  dim3 dimBlock = (32, 1, 1);

  rgb_copy_interleaved<<<dimGrid, dimBlock, 32 * sizeof(pixel)>>>(
      device_pixel_src, device_pixel_cpy);
  hipDeviceSynchronize();

  hipFree(device_pixel_src);
  free(host_pixel);

  return 0;
}