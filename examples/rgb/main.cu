#include "rgb.cu"

int main()
{
    pixel host_pixel[32];
    for (int i = 0; i < 32; i++)
    {
        host_pixel[i] = {i, i, i};
    }
    pixel *device_pixel_src;

    hipMalloc(&device_pixel_src, 32 * sizeof(pixel));

    hipMemcpy(device_pixel_src, host_pixel, 32 * sizeof(pixel), hipMemcpyHostToDevice);

    dim3 dimGrid = (32, 1, 1);
    dim3 dimBlock = (32, 1, 1);

    rgb_smem_interleaved<<<dimGrid, dimBlock, 32 * sizeof(pixel)>>>(device_pixel_src);
    hipDeviceSynchronize();

    hipFree(device_pixel_src);
    free(host_pixel);

    return 0;
}