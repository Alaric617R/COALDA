#include "rgb.cu"
#include <cstdio>

void test_rgb_array()
{
  const dim3 dimGrid = dim3(1, 1, 1);
  const dim3 dimBlock = dim3(TILE_WIDTH, 1, 1);
  int num_pixels = dimGrid.x * dimBlock.x;
  int host_pixel_src[3 * num_pixels];
  int host_pixel_res[3 * num_pixels];
  int *device_pixel_src;
  int *device_pixel_cpy;
  srand(1);
  for (int i = 0; i < num_pixels; i++)
  {
    host_pixel_src[3 * i + 0] = rand() % 256;
    host_pixel_src[3 * i + 1] = rand() % 256;
    host_pixel_src[3 * i + 2] = rand() % 256;
  }
  printf("Host data initialized:\n");
  for (int i = 0; i < num_pixels; i++)
  {
    printf("%d %d %d %d\n", i, host_pixel_src[3 * i + 0], host_pixel_src[3 * i + 1], host_pixel_src[3 * i + 2]);
  }
  hipMalloc(&device_pixel_src, 3 * num_pixels * sizeof(int));
  hipMalloc(&device_pixel_cpy, 3 * num_pixels * sizeof(int));
  hipMemcpy(device_pixel_src, host_pixel_src, 3 * num_pixels * sizeof(int), hipMemcpyHostToDevice);

  rgb_copy_array_interleaved<<<dimGrid, dimBlock>>>(device_pixel_cpy, device_pixel_src);
  hipDeviceSynchronize();

  hipMemcpy(host_pixel_res, device_pixel_cpy, 3 * num_pixels * sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  bool success = true;
  printf("Data after device internal copy:\n");
  for (int i = 0; i < num_pixels; i++)
  {
    printf("%d %d %d %d\n", i, host_pixel_res[3 * i + 0], host_pixel_res[3 * i + 1], host_pixel_res[3 * i + 2]);
    if (host_pixel_res[3 * i + 0] != host_pixel_src[3 * i + 0] ||
        host_pixel_res[3 * i + 1] != host_pixel_src[3 * i + 1] ||
        host_pixel_res[3 * i + 2] != host_pixel_src[3 * i + 2])
    {
      success = false;
      printf("Error on pixel %d\n", i);
    }
  }
  if (success)
  {
    printf("All matched!\n");
  }

  hipFree(device_pixel_src);
  hipFree(device_pixel_cpy);
}

int main()
{
  test_rgb_array();
  return 0;
}