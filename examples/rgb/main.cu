#include "rgb.cu"
#include <cstdio>

void test_rgb_array()
{
  const dim3 dimGrid = dim3(1, 1, 1);
  const dim3 dimBlock = dim3(32, 1, 1);
  int num_pixels = dimGrid.x * dimBlock.x;
  int host_pixel_src[3 * num_pixels];
  int host_pixel_res[3 * num_pixels];
  int *device_pixel_src;
  int *device_pixel_cpy;
  srand(1);
  for (int i = 0; i < num_pixels; i++)
  {
    host_pixel_src[3 * i + 0] = rand() % 256;
    host_pixel_src[3 * i + 1] = rand() % 256;
    host_pixel_src[3 * i + 2] = rand() % 256;
  }
  printf("Host data initialized:\n");
  for (int i = 0; i < num_pixels; i++)
  {
    printf("%d %d %d %d\n", i, host_pixel_src[3 * i + 0], host_pixel_src[3 * i + 1], host_pixel_src[3 * i + 2]);
  }
  hipMalloc(&device_pixel_src, 3 * num_pixels * sizeof(int));
  hipMalloc(&device_pixel_cpy, 3 * num_pixels * sizeof(int));
  hipMemcpy(device_pixel_src, host_pixel_src, 3 * num_pixels * sizeof(int), hipMemcpyHostToDevice);

  rgb_copy_array_interleaved<<<dimGrid, dimBlock>>>(device_pixel_cpy, device_pixel_src);
  hipDeviceSynchronize();

  bool success = true;
  printf("Data after device internal copy:\n");
  for (int i = 0; i < num_pixels; i++)
  {
    printf("%d %d %d %d\n", i, host_pixel_res[3 * i + 0], host_pixel_res[3 * i + 1], host_pixel_res[3 * i + 2]);
    if (host_pixel_res[3 * i + 0] != host_pixel_src[3 * i + 0] ||
        host_pixel_res[3 * i + 1] != host_pixel_src[3 * i + 1] ||
        host_pixel_res[3 * i + 2] != host_pixel_src[3 * i + 2])
    {
      success = false;
      printf("Error on pixel %d\n", i);
    }
  }
  if (success)
  {
    printf("All matched!\n");
  }

  hipFree(device_pixel_src);
  hipFree(device_pixel_cpy);
}

int main()
{
  pixel host_pixel_src[32];
  pixel host_pixel_res[32];
  pixel *device_pixel_src;
  pixel *device_pixel_cpy;

  srand(1);
  for (int i = 0; i < 32; i++)
  {
    host_pixel_src[i] = {rand() % 256, rand() % 256, rand() % 256};
  }

  printf("Host data initialized:\n");
  for (int i = 0; i < 32; i++)
  {
    printf("%d %d %d %d\n", i, host_pixel_src[i].r, host_pixel_src[i].g, host_pixel_src[i].b);
  }

  hipMalloc(&device_pixel_src, 32 * sizeof(pixel));
  hipMalloc(&device_pixel_cpy, 32 * sizeof(pixel));

  hipMemcpy(device_pixel_src, host_pixel_src, 32 * sizeof(pixel), hipMemcpyHostToDevice);

  dim3 dimGrid = dim3(1, 1, 1);
  dim3 dimBlock = dim3(32, 1, 1);

  rgb_copy_struct_interleaved<<<dimGrid, dimBlock>>>(device_pixel_cpy, device_pixel_src);
  hipDeviceSynchronize();

  hipMemcpy(host_pixel_res, device_pixel_cpy, 32 * sizeof(pixel), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  bool success = true;
  printf("Data after device internal copy:\n");
  for (int i = 0; i < 32; i++)
  {
    printf("%d %d %d %d\n", i, host_pixel_res[i].r, host_pixel_res[i].g, host_pixel_res[i].b);
    if (host_pixel_res[i] != host_pixel_src[i])
    {
      success = false;
      printf("Error on pixel %d\n", i);
    }
  }
  if (success)
  {
    printf("All matched!\n");
  }

  hipFree(device_pixel_src);
  hipFree(device_pixel_cpy);

  return 0;
}