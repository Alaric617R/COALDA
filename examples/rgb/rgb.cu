
#include <hip/hip_runtime.h>
struct pixel
{
  int r;
  int g;
  int b;

  bool operator==(const pixel &rhs)
  {
    return (r == rhs.r) && (g == rhs.g) && (b == rhs.b);
  };
  bool operator!=(const pixel &rhs)
  {
    return !(*this == rhs);
  }
};

// Commonly used, intuitive but uncoalesced
__global__ void rgb_copy_array_interleaved(int *pixel_dst, int *pixel_src)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int tmp = 0;
  if (tid < 16)
  {
    pixel_dst[3 * tid + 0] = pixel_src[3 * tid + 0]; // r
    if (threadIdx.x < 10)
    {
      tmp = 1;
    }
    else
    {
      tmp = 2;
    }
    pixel_dst[3 * tid + 1] = pixel_src[3 * tid + 1]; // g
    pixel_dst[3 * tid + 2] = pixel_src[3 * tid + 2]; // b
  }
};

// Desired, coalesced
__global__ void rgb_copy_array_coalesced(int *pixel_dst, int *pixel_src)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  pixel_dst[tid + 0 * blockDim.x] = pixel_src[tid + 0 * blockDim.x]; // r/g/b stride 1
  pixel_dst[tid + 1 * blockDim.x] = pixel_src[tid + 1 * blockDim.x]; // r/g/b stride 2
  pixel_dst[tid + 2 * blockDim.x] = pixel_src[tid + 2 * blockDim.x]; // r/g/b stride 3
};

// Commonly used, intuitive but uncoalesced
__global__ void rgb_copy_struct_interleaved(pixel *pixel_dst, pixel *pixel_src)
{
  pixel_dst[threadIdx.x].r = pixel_src[threadIdx.x].r;
  pixel_dst[threadIdx.x].g = pixel_src[threadIdx.x].g;
  pixel_dst[threadIdx.x].b = pixel_src[threadIdx.x].b;
};

// Desired, coalesced
__global__ void rgb_copy_struct_coalesced(pixel *pixel_dst, pixel *pixel_src)
{
  int *ptr_src = (int *)pixel_src;
  int *ptr_dst = (int *)pixel_dst;
  ptr_dst[threadIdx.x + 0 * blockDim.x] = ptr_src[threadIdx.x + 0 * blockDim.x];
  ptr_dst[threadIdx.x + 1 * blockDim.x] = ptr_src[threadIdx.x + 1 * blockDim.x];
  ptr_dst[threadIdx.x + 2 * blockDim.x] = ptr_src[threadIdx.x + 2 * blockDim.x];
};

__global__ void rgb_copy_struct_whole(pixel *pix_dst, pixel *pixel_src)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  pix_dst[tid] = pixel_src[tid];
};
