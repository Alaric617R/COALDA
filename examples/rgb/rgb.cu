
#include <hip/hip_runtime.h>
struct pixel
{
  int r;
  int g;
  int b;

  bool operator==(const pixel &rhs)
  {
    return (r == rhs.r) && (g == rhs.g) && (b == rhs.b);
  };
  bool operator!=(const pixel &rhs)
  {
    return !(*this == rhs);
  }
};

// Commonly used, intuitive but uncoalesced
__global__ void rgb_copy_interleaved(pixel *pixel_dst, pixel *pixel_src)
{
  pixel_dst[threadIdx.x].r = pixel_src[threadIdx.x].r;
  pixel_dst[threadIdx.x].g = pixel_src[threadIdx.x].g;
  pixel_dst[threadIdx.x].b = pixel_src[threadIdx.x].b;
};

// Desired, coalesced
__global__ void rgb_copy_coalesced(pixel *pixel_dst, pixel *pixel_src)
{
  int *ptr_src = (int *)pixel_src;
  int *ptr_dst = (int *)pixel_dst;
  ptr_dst[threadIdx.x + 0 * blockDim.x] = ptr_src[threadIdx.x + 0 * blockDim.x];
  ptr_dst[threadIdx.x + 1 * blockDim.x] = ptr_src[threadIdx.x + 1 * blockDim.x];
  ptr_dst[threadIdx.x + 2 * blockDim.x] = ptr_src[threadIdx.x + 2 * blockDim.x];
};
