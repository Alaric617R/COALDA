#include "hip/hip_runtime.h"
struct pixel {
  int r;
  int g;
  int b;
};

__global__ void rgb_copy_interleaved(pixel *pixel_dst, pixel *pixel_src) {
  pixel_dst[threadIdx.x].r = pixel_src[threadIdx.x].r;
  pixel_dst[threadIdx.x].g = pixel_src[threadIdx.x].g;
  pixel_dst[threadIdx.x].b = pixel_src[threadIdx.x].b;
};

__global__ void rgb_copy_interleaved(pixel *pixel_dst, pixel *pixel_src) {
  int *ptr_src = (int *)pixel_src;
  int *ptr_dst = (int *)pixel_dst;
  ptr_src[threadIdx.x + 0 * blockDim.x] = ptr_dst[threadIdx.x + 0 * blockDim.x];
  ptr_src[threadIdx.x + 1 * blockDim.x] = ptr_dst[threadIdx.x + 1 * blockDim.x];
  ptr_src[threadIdx.x + 2 * blockDim.x] = ptr_dst[threadIdx.x + 2 * blockDim.x];
};
