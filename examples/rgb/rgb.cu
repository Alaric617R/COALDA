
#include <hip/hip_runtime.h>
struct pixel
{
  int r;
  int g;
  int b;
};

// Commonly used, intuitive but uncoalesced
__global__ void rgb_copy_array_interleaved(int *pixel_dst, int *pixel_src)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int disrupt = tid;
  pixel_dst[3 * tid + 0] = pixel_src[3 * tid + 0]; // r
  pixel_dst[3 * tid + 1] = pixel_src[3 * tid + 1]; // g
  pixel_dst[3 * tid + 2] = pixel_src[3 * tid + 2]; // b
  if (disrupt > 10) disrupt = disrupt * 2;
};

// Desired, coalesced
__global__ void rgb_copy_array_coalesced(int *pixel_dst, int *pixel_src)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  pixel_dst[tid + 0 * blockDim.x] = pixel_src[tid + 0 * blockDim.x]; // r/g/b stride 1
  pixel_dst[tid + 1 * blockDim.x] = pixel_src[tid + 1 * blockDim.x]; // r/g/b stride 2
  pixel_dst[tid + 2 * blockDim.x] = pixel_src[tid + 2 * blockDim.x]; // r/g/b stride 3
};

