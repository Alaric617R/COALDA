
#include <hip/hip_runtime.h>
struct pixel
{
    int r;
    int g;
    int b;
};

__global__ void rgb_smem_interleaved(pixel *pixel_src)
{
    extern __shared__ pixel pixel_dst[];
    // For testing bandwidth, load 4 times repeatedly
    # pragma unroll
    for (size_t i = 0; i < 4; i++)
    {
        pixel_dst[threadIdx.x].r = pixel_src[threadIdx.x].r;
        pixel_dst[threadIdx.x].g = pixel_src[threadIdx.x].g;
        pixel_dst[threadIdx.x].b = pixel_src[threadIdx.x].b;
        __syncthreads();
    }
};

__global__ void rgb_smem_coalesced(pixel *pixel_src)
{
    extern __shared__ pixel pixel_dst[];
    int *ptr_src = (int *)pixel_src;
    int *ptr_dst = (int *)pixel_dst;
    // For testing bandwidth, load 4 times repeatedly
    # pragma unroll
    for (size_t i = 0; i < 4; i++)
    {
        ptr_src[threadIdx.x + 0 * blockDim.x] = ptr_dst[threadIdx.x + 0 * blockDim.x];
        ptr_src[threadIdx.x + 1 * blockDim.x] = ptr_dst[threadIdx.x + 1 * blockDim.x];
        ptr_src[threadIdx.x + 2 * blockDim.x] = ptr_dst[threadIdx.x + 2 * blockDim.x];
        __syncthreads();
    }
};