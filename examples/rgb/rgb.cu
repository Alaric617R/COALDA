
#include <hip/hip_runtime.h>
#define TILE_WIDTH 512

// Commonly used, intuitive but uncoalesced
__global__ void rgb_copy_array_interleaved(int *pixel_dst, int *pixel_src)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int disrupt = tid;
  pixel_dst[3 * tid + 0] = pixel_src[3 * tid + 0]; // r
  pixel_dst[3 * tid + 1] = pixel_src[3 * tid + 1]; // g
  pixel_dst[3 * tid + 2] = pixel_src[3 * tid + 2]; // b
  if (disrupt > 10)
    disrupt = disrupt * 2;
};

// Desired, coalesced
__global__ void rgb_copy_array_coalesced(int *pixel_dst, int *pixel_src)
{
  // IMPORTANT: BUGGY CODE FIX
  int tid = 3 * blockIdx.x * blockDim.x + threadIdx.x;
  pixel_dst[tid + 0 * blockDim.x] = pixel_src[tid + 0 * blockDim.x]; // r/g/b stride 1
  pixel_dst[tid + 1 * blockDim.x] = pixel_src[tid + 1 * blockDim.x]; // r/g/b stride 2
  pixel_dst[tid + 2 * blockDim.x] = pixel_src[tid + 2 * blockDim.x]; // r/g/b stride 3
};

// Commonly used, intuitive but uncoalesced
// Functionality: averaging, new_r/g/b = 0.5 * curr_r/g/b + 0.5 * right_r/g/b
// Assumption: the total number of pixels minus one is a multiple of our TILE_WIDTH and aka blockDim.x
// Pattern: load data into shared memory, compute/read/write only on shared memory, write result in shared memory to global memory
__global__ void rgb_smem_array_interleaved(int *pixel_dst, int *pixel_src, int size)
{

  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH + 3];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
    TODO: uncoalesced, with slightly complicated pattern but still a continuous region of memory
  */
  pixel_smem_src[3 * local_tid + 0] = pixel_src[3 * global_tid + 0]; // r
  pixel_smem_src[3 * local_tid + 1] = pixel_src[3 * global_tid + 1]; // g
  pixel_smem_src[3 * local_tid + 2] = pixel_src[3 * global_tid + 2]; // b
  if (local_tid < 3)                                                 // In this assumption, we always want to load tail
  {
    pixel_smem_src[3 * TILE_WIDTH + local_tid] = pixel_src[3 * TILE_WIDTH + local_tid];
  }

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] = (pixel_smem_src[3 * local_tid + 0] + pixel_smem_src[3 * (local_tid + 1) + 0]) >> 1;
  pixel_smem_dst[3 * local_tid + 1] = (pixel_smem_src[3 * local_tid + 1] + pixel_smem_src[3 * (local_tid + 1) + 1]) >> 1;
  pixel_smem_dst[3 * local_tid + 2] = (pixel_smem_src[3 * local_tid + 2] + pixel_smem_src[3 * (local_tid + 1) + 2]) >> 1;

  /*
    Write result to destination
    TODO: uncoalesced
  */
  __syncthreads();
  pixel_dst[3 * global_tid + 0] = pixel_smem_dst[3 * local_tid + 0]; // r
  __syncthreads();
  pixel_dst[3 * global_tid + 1] = pixel_smem_dst[3 * local_tid + 1]; // g
  __syncthreads();
  pixel_dst[3 * global_tid + 2] = pixel_smem_dst[3 * local_tid + 2]; // b
};

__global__ void rgb_increase_brightness(int *pixel_dst, int *pixel_src,
                                        int size, float factor) {
  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
  */
  pixel_smem_src[3 * local_tid + 0] = pixel_src[3 * global_tid + 0];  // r
  __syncthreads();
  pixel_smem_src[3 * local_tid + 1] = pixel_src[3 * global_tid + 1];  // g
  __syncthreads();
  pixel_smem_src[3 * local_tid + 2] = pixel_src[3 * global_tid + 2];  // b
  __syncthreads();

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 0])));
  pixel_smem_dst[3 * local_tid + 1] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 1])));
  pixel_smem_dst[3 * local_tid + 2] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 2])));
  __syncthreads();

  /*
    Write result to destination
  */
  pixel_dst[3 * global_tid + 0] = pixel_smem_dst[3 * local_tid + 0];  // r
  __syncthreads();
  pixel_dst[3 * global_tid + 1] = pixel_smem_dst[3 * local_tid + 1];  // g
  __syncthreads();
  pixel_dst[3 * global_tid + 2] = pixel_smem_dst[3 * local_tid + 2];  // b
};

__global__ void rgb_increase_brightness_coalesced(int *pixel_dst,
                                                  int *pixel_src, int size,
                                                  float factor) {
  /*
    Declaration
  */
  int global_tid = 3 * blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
  */
  pixel_smem_src[local_tid + 0 * blockDim.x] =
      pixel_src[global_tid + 0 * blockDim.x];  // r
  __syncthreads();
  pixel_smem_src[local_tid + 1 * blockDim.x] =
      pixel_src[global_tid + 1 * blockDim.x];  // g
  __syncthreads();
  pixel_smem_src[local_tid + 2 * blockDim.x] =
      pixel_src[global_tid + 2 * blockDim.x];  // b
  __syncthreads();

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 0])));
  pixel_smem_dst[3 * local_tid + 1] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 1])));
  pixel_smem_dst[3 * local_tid + 2] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 2])));
  __syncthreads();

  /*
    Write result to destination
  */
  pixel_dst[global_tid + 0 * blockDim.x] =
      pixel_smem_dst[local_tid + 0 * blockDim.x];  // r
  __syncthreads();
  pixel_dst[global_tid + 1 * blockDim.x] =
      pixel_smem_dst[local_tid + 1 * blockDim.x];  // g
  __syncthreads();
  pixel_dst[global_tid + 2 * blockDim.x] =
      pixel_smem_dst[local_tid + 2 * blockDim.x];  // b
};

__global__ void rgb_increase_brightness_unroll(int *pixel_dst, int *pixel_src,
                                        int size, float factor) {
  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[4 * 3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[4 * 3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
  */
  pixel_smem_src[4 * 3 * local_tid + 0] = pixel_src[4 * 3 * global_tid + 0];  // r
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 1] = pixel_src[4 * 3 * global_tid + 1];  // g
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 2] = pixel_src[4 * 3 * global_tid + 2];  // b
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 3] = pixel_src[4 * 3 * global_tid + 3];  // r
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 4] = pixel_src[4 * 3 * global_tid + 4];  // g
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 5] = pixel_src[4 * 3 * global_tid + 5];  // b
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 6] = pixel_src[4 * 3 * global_tid + 6];  // r
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 7] = pixel_src[4 * 3 * global_tid + 7];  // g
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 8] = pixel_src[4 * 3 * global_tid + 8];  // b
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 9] = pixel_src[4 * 3 * global_tid + 9];  // r
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 10] = pixel_src[4 * 3 * global_tid + 10];  // g
  __syncthreads();
  pixel_smem_src[4 * 3 * local_tid + 11] = pixel_src[4 * 3 * global_tid + 11];  // b
  __syncthreads();

  /*
    Computation
  */
  pixel_smem_dst[4 * 3 * local_tid + 0] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 0])));
  pixel_smem_dst[4 * 3 * local_tid + 1] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 1])));
  pixel_smem_dst[4 * 3 * local_tid + 2] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 2])));
  pixel_smem_dst[4 * 3 * local_tid + 3] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 3])));
  pixel_smem_dst[4 * 3 * local_tid + 4] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 4])));
  pixel_smem_dst[4 * 3 * local_tid + 5] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 5])));
  pixel_smem_dst[4 * 3 * local_tid + 6] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 6])));
  pixel_smem_dst[4 * 3 * local_tid + 7] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 7])));
  pixel_smem_dst[4 * 3 * local_tid + 8] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 8])));
  pixel_smem_dst[4 * 3 * local_tid + 9] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 9])));
  pixel_smem_dst[4 * 3 * local_tid + 10] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 10])));
  pixel_smem_dst[4 * 3 * local_tid + 11] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 11])));
  __syncthreads();

  /*
    Write result to destination
  */
  pixel_dst[4 * 3 * global_tid + 0] = pixel_smem_dst[4 * 3 * local_tid + 0];  // r
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 1] = pixel_smem_dst[4 * 3 * local_tid + 1];  // g
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 2] = pixel_smem_dst[4 * 3 * local_tid + 2];  // b
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 3] = pixel_smem_dst[4 * 3 * local_tid + 3];  // r
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 4] = pixel_smem_dst[4 * 3 * local_tid + 4];  // g
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 5] = pixel_smem_dst[4 * 3 * local_tid + 5];  // b
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 6] = pixel_smem_dst[4 * 3 * local_tid + 6];  // r
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 7] = pixel_smem_dst[4 * 3 * local_tid + 7];  // g
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 8] = pixel_smem_dst[4 * 3 * local_tid + 8];  // b
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 9] = pixel_smem_dst[4 * 3 * local_tid + 9];  // r
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 10] = pixel_smem_dst[4 * 3 * local_tid + 10];  // g
  __syncthreads();
  pixel_dst[4 * 3 * global_tid + 11] = pixel_smem_dst[4 * 3 * local_tid + 11];  // b
  __syncthreads();
};

__global__ void rgb_increase_brightness_coalesced_unroll(int *pixel_dst,
                                                  int *pixel_src, int size,
                                                  float factor) {
  /*
    Declaration
  */
  int global_tid = 4 * 3 * blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[4 * 3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[4 * 3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
  */
  pixel_smem_src[local_tid + 0 * blockDim.x] =
      pixel_src[global_tid + 0 * blockDim.x];  // r
  __syncthreads();
  pixel_smem_src[local_tid + 1 * blockDim.x] =
      pixel_src[global_tid + 1 * blockDim.x];  // g
  __syncthreads();
  pixel_smem_src[local_tid + 2 * blockDim.x] =
      pixel_src[global_tid + 2 * blockDim.x];  // b
  __syncthreads();
  pixel_smem_src[local_tid + 3 * blockDim.x] =
      pixel_src[global_tid + 3 * blockDim.x];  // r
  __syncthreads();
  pixel_smem_src[local_tid + 4 * blockDim.x] =
      pixel_src[global_tid + 4 * blockDim.x];  // g
  __syncthreads();
  pixel_smem_src[local_tid + 5 * blockDim.x] =
      pixel_src[global_tid + 5 * blockDim.x];  // b
  __syncthreads();
  pixel_smem_src[local_tid + 6 * blockDim.x] =
      pixel_src[global_tid + 6 * blockDim.x];  // r
  __syncthreads();
  pixel_smem_src[local_tid + 7 * blockDim.x] =
      pixel_src[global_tid + 7 * blockDim.x];  // g
  __syncthreads();
  pixel_smem_src[local_tid + 8 * blockDim.x] =
      pixel_src[global_tid + 8 * blockDim.x];  // b
  __syncthreads();
  pixel_smem_src[local_tid + 9 * blockDim.x] =
      pixel_src[global_tid + 9 * blockDim.x];  // r
  __syncthreads();
  pixel_smem_src[local_tid + 10 * blockDim.x] =
      pixel_src[global_tid + 10 * blockDim.x];  // g
  __syncthreads();
  pixel_smem_src[local_tid + 11 * blockDim.x] =
      pixel_src[global_tid + 11 * blockDim.x];  // b
  __syncthreads();

  /*
    Computation
  */
  pixel_smem_dst[4 * 3 * local_tid + 0] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 0])));
  pixel_smem_dst[4 * 3 * local_tid + 1] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 1])));
  pixel_smem_dst[4 * 3 * local_tid + 2] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 2])));
  pixel_smem_dst[4 * 3 * local_tid + 3] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 3])));
  pixel_smem_dst[4 * 3 * local_tid + 4] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 4])));
  pixel_smem_dst[4 * 3 * local_tid + 5] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 5])));
  pixel_smem_dst[4 * 3 * local_tid + 6] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 6])));
  pixel_smem_dst[4 * 3 * local_tid + 7] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 7])));
  pixel_smem_dst[4 * 3 * local_tid + 8] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 8])));
  pixel_smem_dst[4 * 3 * local_tid + 9] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 9])));
  pixel_smem_dst[4 * 3 * local_tid + 10] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 10])));
  pixel_smem_dst[4 * 3 * local_tid + 11] =
      min(255, (int)(factor * (pixel_smem_src[4 * 3 * local_tid + 11])));
  __syncthreads();

  /*
    Write result to destination
  */
  pixel_dst[global_tid + 0 * blockDim.x] =
      pixel_smem_dst[local_tid + 0 * blockDim.x];  // r
  __syncthreads();
  pixel_dst[global_tid + 1 * blockDim.x] =
      pixel_smem_dst[local_tid + 1 * blockDim.x];  // g
  __syncthreads();
  pixel_dst[global_tid + 2 * blockDim.x] =
      pixel_smem_dst[local_tid + 2 * blockDim.x];  // b
  __syncthreads();
  pixel_dst[global_tid + 3 * blockDim.x] =
      pixel_smem_dst[local_tid + 3 * blockDim.x];  // r
  __syncthreads();
  pixel_dst[global_tid + 4 * blockDim.x] =
      pixel_smem_dst[local_tid + 4 * blockDim.x];  // g
  __syncthreads();
  pixel_dst[global_tid + 5 * blockDim.x] =
      pixel_smem_dst[local_tid + 5 * blockDim.x];  // b
  __syncthreads();
  pixel_dst[global_tid + 6 * blockDim.x] =
      pixel_smem_dst[local_tid + 6 * blockDim.x];  // r
  __syncthreads();
  pixel_dst[global_tid + 7 * blockDim.x] =
      pixel_smem_dst[local_tid + 7 * blockDim.x];  // g
  __syncthreads();
  pixel_dst[global_tid + 8 * blockDim.x] =
      pixel_smem_dst[local_tid + 8 * blockDim.x];  // b
  __syncthreads();
  pixel_dst[global_tid + 9 * blockDim.x] =
      pixel_smem_dst[local_tid + 9 * blockDim.x];  // r
  __syncthreads();
  pixel_dst[global_tid + 10 * blockDim.x] =
      pixel_smem_dst[local_tid + 10 * blockDim.x];  // g
  __syncthreads();
  pixel_dst[global_tid + 11 * blockDim.x] =
      pixel_smem_dst[local_tid + 11 * blockDim.x];  // b
  __syncthreads();
};
