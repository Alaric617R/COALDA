
#include <hip/hip_runtime.h>
#define TILE_WIDTH 32

// Commonly used, intuitive but uncoalesced
__global__ void rgb_copy_array_interleaved(int *pixel_dst, int *pixel_src)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int disrupt = tid;
  pixel_dst[3 * tid + 0] = pixel_src[3 * tid + 0]; // r
  pixel_dst[3 * tid + 1] = pixel_src[3 * tid + 1]; // g
  pixel_dst[3 * tid + 2] = pixel_src[3 * tid + 2]; // b
  if (disrupt > 10)
    disrupt = disrupt * 2;
};

// Desired, coalesced
__global__ void rgb_copy_array_coalesced(int *pixel_dst, int *pixel_src)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  pixel_dst[tid + 0 * blockDim.x] = pixel_src[tid + 0 * blockDim.x]; // r/g/b stride 1
  pixel_dst[tid + 1 * blockDim.x] = pixel_src[tid + 1 * blockDim.x]; // r/g/b stride 2
  pixel_dst[tid + 2 * blockDim.x] = pixel_src[tid + 2 * blockDim.x]; // r/g/b stride 3
};

// Commonly used, intuitive but uncoalesced
// Functionality: averaging, new_r/g/b = 0.5 * curr_r/g/b + 0.5 * right_r/g/b
// Assumption: the total number of pixels is a multiple of our TILE_WIDTH and aka blockDim.x
// Pattern: load data into shared memory, compute/read/write only on shared memory, write result in shared memory to global memory
__global__ void rgb_smem_array_interleaved(int *pixel_dst, int *pixel_src, int size)
{

  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH + 3];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];
  __shared__ bool valid_tail;

  /*
    Check if we want to load extra tail element
    Note we only need one thread for computing this
  */
  if (local_tid == TILE_WIDTH - 1)
  {
    valid_tail = (global_tid + 1 < size); // e.g. global_tid = 31, size = 32, invalid tail; global_tid = 31, size = 33, valid tail
  }

  /*
    Read input pixels to shared memory
    TODO: uncoalesced, with slightly complicated pattern but still a continuous region of memory
  */
  pixel_smem_src[3 * local_tid + 0] = pixel_src[3 * global_tid + 0]; // r
  pixel_smem_src[3 * local_tid + 1] = pixel_src[3 * global_tid + 1]; // g
  pixel_smem_src[3 * local_tid + 2] = pixel_src[3 * global_tid + 2]; // b
  if (valid_tail && local_tid < 3)
  {
    pixel_smem_src[3 * TILE_WIDTH + local_tid] = pixel_src[3 * TILE_WIDTH + local_tid];
  }

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] = pixel_smem_src[3 * local_tid + 0] + pixel_smem_dst[3 * (local_tid + 1) + 0];
  pixel_smem_dst[3 * local_tid + 1] = pixel_smem_src[3 * local_tid + 1] + pixel_smem_dst[3 * (local_tid + 1) + 1];
  pixel_smem_dst[3 * local_tid + 2] = pixel_smem_src[3 * local_tid + 2] + pixel_smem_dst[3 * (local_tid + 1) + 2];

  /*
    Write result to destination
    TODO: uncoalesced
  */
  pixel_dst[3 * global_tid + 0] = pixel_smem_src[3 * local_tid + 0]; // r
  pixel_dst[3 * global_tid + 1] = pixel_smem_src[3 * local_tid + 1]; // g
  pixel_dst[3 * global_tid + 2] = pixel_smem_src[3 * local_tid + 2]; // b
};