
#include <hip/hip_runtime.h>
#define TILE_WIDTH 1024

// Commonly used, intuitive but uncoalesced
__global__ void rgb_copy_array_interleaved(int *pixel_dst, int *pixel_src)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int disrupt = tid;
  pixel_dst[3 * tid + 0] = pixel_src[3 * tid + 0]; // r
  pixel_dst[3 * tid + 1] = pixel_src[3 * tid + 1]; // g
  pixel_dst[3 * tid + 2] = pixel_src[3 * tid + 2]; // b
  if (disrupt > 10)
    disrupt = disrupt * 2;
};

// Desired, coalesced
__global__ void rgb_copy_array_coalesced(int *pixel_dst, int *pixel_src)
{
  // IMPORTANT: BUGGY CODE FIX
  int tid = 3 * blockIdx.x * blockDim.x + threadIdx.x;
  pixel_dst[tid + 0 * blockDim.x] = pixel_src[tid + 0 * blockDim.x]; // r/g/b stride 1
  pixel_dst[tid + 1 * blockDim.x] = pixel_src[tid + 1 * blockDim.x]; // r/g/b stride 2
  pixel_dst[tid + 2 * blockDim.x] = pixel_src[tid + 2 * blockDim.x]; // r/g/b stride 3
};

// Commonly used, intuitive but uncoalesced
// Functionality: averaging, new_r/g/b = 0.5 * curr_r/g/b + 0.5 * right_r/g/b
// Assumption: the total number of pixels minus one is a multiple of our TILE_WIDTH and aka blockDim.x
// Pattern: load data into shared memory, compute/read/write only on shared memory, write result in shared memory to global memory
__global__ void rgb_smem_array_interleaved(int *pixel_dst, int *pixel_src, int size)
{

  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH + 3];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
    TODO: uncoalesced, with slightly complicated pattern but still a continuous region of memory
  */
  pixel_smem_src[3 * local_tid + 0] = pixel_src[3 * global_tid + 0]; // r
  pixel_smem_src[3 * local_tid + 1] = pixel_src[3 * global_tid + 1]; // g
  pixel_smem_src[3 * local_tid + 2] = pixel_src[3 * global_tid + 2]; // b
  if (local_tid < 3)                                                 // In this assumption, we always want to load tail
  {
    pixel_smem_src[3 * TILE_WIDTH + local_tid] = pixel_src[3 * TILE_WIDTH + local_tid];
  }

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] = (pixel_smem_src[3 * local_tid + 0] + pixel_smem_src[3 * (local_tid + 1) + 0]) >> 1;
  pixel_smem_dst[3 * local_tid + 1] = (pixel_smem_src[3 * local_tid + 1] + pixel_smem_src[3 * (local_tid + 1) + 1]) >> 1;
  pixel_smem_dst[3 * local_tid + 2] = (pixel_smem_src[3 * local_tid + 2] + pixel_smem_src[3 * (local_tid + 1) + 2]) >> 1;

  /*
    Write result to destination
    TODO: uncoalesced
  */
  pixel_dst[3 * global_tid + 0] = pixel_smem_dst[3 * local_tid + 0]; // r
  pixel_dst[3 * global_tid + 1] = pixel_smem_dst[3 * local_tid + 1]; // g
  pixel_dst[3 * global_tid + 2] = pixel_smem_dst[3 * local_tid + 2]; // b
};