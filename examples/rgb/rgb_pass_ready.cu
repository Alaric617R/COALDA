
#include <hip/hip_runtime.h>
#define TILE_WIDTH 512

__global__ void rgb_increase_brightness_pass_ready(int *pixel_dst,
                                                   int *pixel_src, int size,
                                                   float factor) {
  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
  */
  pixel_smem_src[3 * local_tid + 0] = pixel_src[3 * global_tid + 0];  // r
  __syncthreads();
  pixel_smem_src[3 * local_tid + 1] = pixel_src[3 * global_tid + 1];  // g
  __syncthreads();
  pixel_smem_src[3 * local_tid + 2] = pixel_src[3 * global_tid + 2];  // b
  __syncthreads();

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 0])));
  pixel_smem_dst[3 * local_tid + 1] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 1])));
  pixel_smem_dst[3 * local_tid + 2] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 2])));
  __syncthreads();

  /*
    Write result to destination
  */
  pixel_dst[3 * global_tid + 0] = pixel_smem_dst[3 * local_tid + 0];  // r
  __syncthreads();
  pixel_dst[3 * global_tid + 1] = pixel_smem_dst[3 * local_tid + 1];  // g
  __syncthreads();
  pixel_dst[3 * global_tid + 2] = pixel_smem_dst[3 * local_tid + 2];  // b
};

__global__ void rgb_increase_brightness_pass_ready_coalesced(int *pixel_dst,
                                                             int *pixel_src,
                                                             int size,
                                                             float factor) {
  /*
    Declaration
  */
  int global_tid = 3 * blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
  */
  pixel_smem_src[local_tid + 0 * blockDim.x] =
      pixel_src[global_tid + 0 * blockDim.x];  // r
  __syncthreads();
  pixel_smem_src[local_tid + 1 * blockDim.x] =
      pixel_src[global_tid + 1 * blockDim.x];  // g
  __syncthreads();
  pixel_smem_src[local_tid + 2 * blockDim.x] =
      pixel_src[global_tid + 2 * blockDim.x];  // b
  __syncthreads();

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 0])));
  pixel_smem_dst[3 * local_tid + 1] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 1])));
  pixel_smem_dst[3 * local_tid + 2] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 2])));
  __syncthreads();

  /*
    Write result to destination
  */
  pixel_dst[global_tid + 0 * blockDim.x] =
      pixel_smem_dst[local_tid + 0 * blockDim.x];  // r
  __syncthreads();
  pixel_dst[global_tid + 1 * blockDim.x] =
      pixel_smem_dst[local_tid + 1 * blockDim.x];  // g
  __syncthreads();
  pixel_dst[global_tid + 2 * blockDim.x] =
      pixel_smem_dst[local_tid + 2 * blockDim.x];  // b
};