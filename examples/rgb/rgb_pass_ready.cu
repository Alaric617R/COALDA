
#include <hip/hip_runtime.h>
#define TILE_WIDTH 512

__global__ void rgb_increase_brightness_pass_ready(int *pixel_dst, int *pixel_src,
                                        int size, float factor) {
  /*
    Declaration
  */
  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int local_tid = threadIdx.x;
  __shared__ int pixel_smem_src[3 * TILE_WIDTH];
  __shared__ int pixel_smem_dst[3 * TILE_WIDTH];

  /*
    Read input pixels to shared memory
  */
  pixel_smem_src[3 * local_tid + 0] = pixel_src[3 * global_tid + 0];  // r
  __syncthreads();
  pixel_smem_src[3 * local_tid + 1] = pixel_src[3 * global_tid + 1];  // g
  __syncthreads();
  pixel_smem_src[3 * local_tid + 2] = pixel_src[3 * global_tid + 2];  // b
  __syncthreads();

  /*
    Computation
  */
  pixel_smem_dst[3 * local_tid + 0] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 0])));
  pixel_smem_dst[3 * local_tid + 1] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 1])));
  pixel_smem_dst[3 * local_tid + 2] =
      min(255, (int)(factor * (pixel_smem_src[3 * local_tid + 2])));
  __syncthreads();

  /*
    Write result to destination
  */
  pixel_dst[3 * global_tid + 0] = pixel_smem_dst[3 * local_tid + 0];  // r
  __syncthreads();
  pixel_dst[3 * global_tid + 1] = pixel_smem_dst[3 * local_tid + 1];  // g
  __syncthreads();
  pixel_dst[3 * global_tid + 2] = pixel_smem_dst[3 * local_tid + 2];  // b
};