#include <cstdio>

#include "ppm_utils.h"
#include "rgb_pass_ready_coalesced.cu"

#define TILE_WIDTH 512

void test_increase_brightness_pass_ready_coalesced() {
  /*
    Declaration and configuration
  */
  int width, height;
  int *host_pixel_src = read_ppm("images/1.ppm", width, height);
  long long total_pixel = width * height;
  int num_block = (total_pixel + TILE_WIDTH - 1) / TILE_WIDTH;
  const dim3 dimGrid = dim3(num_block, 1, 1);
  const dim3 dimBlock = dim3(TILE_WIDTH, 1, 1);
  int num_pixels = width * height;
  int host_pixel_res[3 * num_pixels];
  int *device_pixel_src;
  int *device_pixel_cpy;

  /*
    Allocate host and device memory
  */
  hipMalloc(&device_pixel_src, 3 * num_pixels * sizeof(int));
  hipMalloc(&device_pixel_cpy, 3 * num_pixels * sizeof(int));

  /*
    Data movement and kernel launch
  */
  hipMemcpy(device_pixel_src, host_pixel_src, 3 * num_pixels * sizeof(int),
             hipMemcpyHostToDevice);
  rgb_increase_brightness_pass_ready_coalesced<<<dimGrid, dimBlock>>>(
      device_pixel_cpy, device_pixel_src, width * height, 1.8);
  hipDeviceSynchronize();
  hipMemcpy(host_pixel_res, device_pixel_cpy, 3 * num_pixels * sizeof(int),
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  /*
    Write result to output file
  */
  write_ppm("images/1_modified.ppm", host_pixel_res, width, height);

  /*
    Free host and device memory
  */
  delete[] host_pixel_src;
  hipFree(device_pixel_src);
  hipFree(device_pixel_cpy);
}

int main() {
  test_increase_brightness_pass_ready_coalesced();
  return 0;
}